#include "hip/hip_runtime.h"
/*****************************************************************************/
//
// MD code for GPU based on a modified version of the CHARMM force field, with
// reaction force field (RF) and Particle Mesh Ewald (PME)
//
// Global Computing Lab, University of Delaware
// Author(s): Narayan Ganesan (narayan.ganesan.8@gmail.com)
//            Joseph E. Davis
//            Michela Taufer (taufer@udel.edu)
// Contact(s): Michela Taufer (taufer@udel.edu)
// Reference(s):
//
/*****************************************************************************/
#include "globals.h"

#include <hip/hip_vector_types.h>  // required for float2

// Struct alignment is handled differently between the CUDA compiler and other
// compilers (e.g. GCC, MS Visual C++ .NET)
#ifdef __HIPCC__
#   define ALIGN(x)  __align__(x)
#else
#   if defined(_MSC_VER) && (_MSC_VER >= 1300)
    //Visual C++ .NET and later
#      define ALIGN(x) __declspec(align(x)) 
#   else
#      if defined(__GNUC__)
// GCC
#         define ALIGN(x)  __attribute__ ((aligned (x)))
#      else
// all other compilers
#         define ALIGN(x) 
#      endif //defined(__GNUC__)
#   endif //defined(_MSC_VER) && (_MSC_VER >= 1300)
#endif //__HIPCC__

//Somehow in emulation mode the code won't compile Mac OS X 1.1 CUDA SDK when
//the operators below make use of references (compiler bug?). So instead we
//compile the code to pass everything through the stack. Slower, but works.
//I am not sure how the Linux CUDA SDK will behave, so currently when I detect
//Microsoft's Visual C++.NET I always allow it to use references.
#if !defined(__DEVICE_EMULATION__) || (defined(_MSC_VER) && (_MSC_VER >= 1300))
#   define REF(x) &x
#   define ARRAYREF(x,y) (&x)[y]
#else
#   define REF(x) x
#   define ARRAYREF(x,y) x[y]
#endif

/**
 * A complex number type for use with CUDA, single precision accuracy.
 * This is deliberately designed to use few C++ features in order to work
 * with most CUDA SDK versions. It is friendlier to use than the hipComplex
 * type because it provides more operator overloads.
 * The class should work in host code and in device code and also in
 * emulation mode.
 * Also this has been tested on any OS that the CUDA SDK is available for.
 */

#define LOWER_BITMASK  0x000FFFFF
#define UPPER_BITMASK  0xFFF00000

#define pack_type_and_atomid(id)  ((tex1Dfetch(textype, (id))<<20)|(id))
#define unpack_typeid(id)         ((id)>>20)
#define unpack_atomid(id)         ((id)&LOWER_BITMASK)


__forceinline__ __device__ float2 lower_float2(float4 a){
	float2 result = {a.y, a.z};
	return result;
}

//add float3 numbers..
__forceinline__ __device__ float3 operator+(const float3 REF(a),
                                            const float3 REF(b)){

	float3 result = { __my_fadd(a.x, b.x),
	                  __my_fadd(a.y, b.y),
	                  __my_fadd(a.z, b.z)};
	return result;
}

//subtract float3 numbers..
__forceinline__ __device__ float3 operator-(const float3 REF(a),
                                            const float3 REF(b)){
	float3 result = { __my_fadd(a.x, -b.x),
	                  __my_fadd(a.y, -b.y),
	                  __my_fadd(a.z, -b.z)};
	return result;
}

//multiply by scalar..
__forceinline__ __device__ float3 operator*(const float3 REF(a),
                                            const float REF(b)){
	float3 result = { __my_fmul(a.x, b),
	                  __my_fmul(a.y, b),
	                  __my_fmul(a.z, b)};
	return result;
}

//divide float3 by scalar..
__forceinline__ __device__ float3 operator/(const float3 REF(a),
                                            const float REF(b)){
	float3 result = { __fdividef(a.x, b),
	                  __fdividef(a.y, b),
	                  __fdividef(a.z, b)};
	return result;
}

//divide float3 by another float3 elementwise..
__forceinline__ __device__ float3 operator/(const float3 REF(a),
                                            const float3 REF(b)){
	float3 result = { __fdividef(a.x, b.x),
	                  __fdividef(a.y, b.y),
	                  __fdividef(a.z, b.z)};
	return result;
}

//vector dot product of 2 float3 variables....
__forceinline__ __device__ float operator%(const float3 REF(a),
                                           const float3 REF(b)){
	float result = { __my_add3(__my_fmul(a.x, b.x),
	                           __my_fmul(a.y, b.y),
	                           __my_fmul(a.z, b.z))};
	return result;
}

//multiply 2 float3 numbers elementwise...
__forceinline__ __device__ float3 operator*(const float3 REF(a),
                                            const float3 REF(b)){
	float3 result = { __my_fmul(a.x, b.x),
	                  __my_fmul(a.y, b.y),
	                  __my_fmul(a.z, b.z)};
	return result;
}

//vector cross product of 2 float3 variables....
__forceinline__ __device__ float3 operator^(const float3 REF(a),
                                            const float3 REF(b)){
	float3 result = { __my_fadd(__my_fmul(a.y, b.z), -__my_fmul(a.z, b.y)), 
	                  __my_fadd(__my_fmul(a.z, b.x), -__my_fmul(a.x, b.z)),
	                  __my_fadd(__my_fmul(a.x, b.y), -__my_fmul(a.y, b.x))};
	return result;
}

__forceinline__ __host__ __device__ float sum(const float3 REF(a)){
	return (a.x + a.y + a.z);
}

__forceinline__ __device__ void operator+=(float3 REF(a), const float3 REF(b)){
	a.x = __my_fadd(a.x, b.x);
	a.y = __my_fadd(a.y, b.y);
	a.z = __my_fadd(a.z, b.z);
	return;
}

__forceinline__ __device__ void operator-=(float3 REF(a), const float3 REF(b)){
	a.x = __my_fadd(a.x, -b.x);
	a.y = __my_fadd(a.y, -b.y);
	a.z = __my_fadd(a.z, -b.z);
	return;
}

__forceinline__ __device__ int3 int4_to_int3(const int4 REF(a)){
	int3 result = {a.x, a.y, a.z};
	return result;
}

__forceinline__ __host__ __device__ void operator+=(int2 &a, int2 b){
	a.x += b.x; a.y += b.y;
}


//convert float4 to float3....
__forceinline__ __device__ float3 float4_to_float3(const float4 REF(a)){
	float3 result = {a.x, a.y, a.z};
	return result;
}

//pad float3 to float4....
__forceinline__ __device__ float4 float3_to_float4(const float3 REF(a)){
	float4 result = {a.x, a.y, a.z, 1.0f};
	return result;
}

//add 2 float4 variables..
__forceinline__ __device__ float4 operator+(const float4 REF(a),
                                            const float4 REF(b)){
	float4 result = { __my_fadd(a.x, b.x), __my_fadd(a.y, b.y),
	                  __my_fadd(a.z, b.z), __my_fadd(a.w, b.w)};
	return result;
}

//multiply float4 by scalar..
__forceinline__ __device__ float4 operator*(const float4 REF(a),
                                            const float REF(b)){
	float4 result = { __my_fmul(a.x, b), __my_fmul(a.y, b),
	                  __my_fmul(a.z, b), __my_fmul(a.w, b)};
	return result;
}

//divide float4 by scalar..
__forceinline__ __device__ float4 operator/(const float4 REF(a),
                                            const float REF(b)){
	float4 result = { __fdividef(a.x, b), __fdividef(a.y, b),
	                  __fdividef(a.z, b), __fdividef(a.w, b)};
	return result;
}

//divide float4 by float4 elementwise..
__forceinline__ __device__ float4 operator/(const float4 REF(a),
                                            const float4 REF(b)){
	float4 result = { __fdividef(a.x, b.x), __fdividef(a.y, b.y),
	                  __fdividef(a.z, b.z), __fdividef(a.w, b.w)};
	return result;
}

//subtract 2 float4 numbers..
__forceinline__ __device__ float4 operator-(const float4 REF(a),
                                            const float4 REF(b)){
	float4 result = { __my_fadd(a.x, -b.x), __my_fadd(a.y, -b.y),
	                  __my_fadd(a.z, -b.z), __my_fadd(a.w, -b.w)};
	return result;
}

//multiply 2 float4 numbers elementwise...
__forceinline__ __device__ float4 operator*(const float4 REF(a),
                                            const float4 REF(b)){
	float4 result = { __my_fmul(a.x, b.x), __my_fmul(a.y, b.y),
	                  __my_fmul(a.z, b.z), __my_fmul(a.w, b.w)};
	return result;
}

__forceinline__ __device__ float4 To_float4(const float3 REF(a),
                                            const float REF(b)){
	float4 result = { a.x, a.y, a.z, b};
	return result;
}

__forceinline__ __device__ float4 To_float4(const float2 REF(a),
                                            const float2 REF(b)){
	float4 result = { a.x, a.y, b.x, b.y};
	return result;
}

//add 2 float4 numbers and assign the result to the first
__forceinline__ __device__ void operator+=(float4 REF(a), const float4 REF(b)){
	a.x = __my_fadd(a.x, b.x);
	a.y = __my_fadd(a.y, b.y);
	a.z = __my_fadd(a.z, b.z);
	a.w = __my_fadd(a.w, b.w);
	return;
}

//add float4 and float3 numbers and assign the result to the first
__forceinline__ __device__ void operator+=(float4 REF(a), const float3 REF(b)){
	a.x = __my_fadd(a.x, b.x);
	a.y = __my_fadd(a.y, b.y);
	a.z = __my_fadd(a.z, b.z);
	return;
}

__forceinline__ __device__ void operator-=(float4 REF(a), const float4 REF(b)){
	a.x = __my_fadd(a.x, -b.x);
	a.y = __my_fadd(a.y, -b.y);
	a.z = __my_fadd(a.z, -b.z);
	a.w = __my_fadd(a.w, -b.w);
	return;
}

__forceinline__ __host__ __device__ float sum(const float4 REF(a)){
	return (a.x + a.y + a.z + a.w);
}

/*
inline __device__ float4 make_float4(float &x, float &y, float &z, float &w){
	float4 result = {x, y, z, w};
	return result;
}
*/

// float3 to float4 assignment..
//__device__ void operator=(const float4 REF(a), const float3 REF(b)){
//	//the last component is unchanged in float3 to float4 assignment...
//	float4 result = { b.x, b.y, b.z, a.w};
//	return result;
//}


//a possible alternative to a hipfftComplex constructor
HOSTDEVICE hipfftComplex make_cufftComplex(float a, float b){
	hipfftComplex res = {a, b};
	return res;
}

namespace constants{
	const hipfftComplex zero = make_cufftComplex(0.0f, 0.0f);
	const hipfftComplex one  = make_cufftComplex(1.0f, 0.0f);
	const hipfftComplex I    = make_cufftComplex(0.0f, 1.0f);
};

//add complex numbers
HOSTDEVICE hipfftComplex operator+(const hipfftComplex REF(a),
                                  const hipfftComplex REF(b)){
	hipfftComplex result = { a.x + b.x, a.y  + b.y };
	return result;
}

//add scalar to complex
HOSTDEVICE hipfftComplex operator+(const hipfftComplex REF(a),
                                  const float REF(b)){
	hipfftComplex result = { a.x + b, a.y };
	return result;
}

//add complex to scalar
HOSTDEVICE hipfftComplex operator+(const float REF(a),
                                  const hipfftComplex REF(b)){
	hipfftComplex result = { a + b.x, b.y };
	return result;
}

//subtract complex numbers
HOSTDEVICE hipfftComplex operator-(const hipfftComplex REF(a),
                                  const hipfftComplex REF(b)){
	hipfftComplex result = { a.x - b.x, a.y  - b.y };
	return result;
}

//negate a complex number
HOSTDEVICE hipfftComplex operator-(const hipfftComplex REF(a)){
	hipfftComplex result = { -a.x, -a.y };
	return result;
}

//subtract scalar from complex
HOSTDEVICE hipfftComplex operator-(const hipfftComplex REF(a),
                                  const float REF(b)){
	hipfftComplex result = { a.x - b, a.y };
	return result;
}

//subtract complex from scalar
HOSTDEVICE hipfftComplex operator-(const float REF(a),
                                  const hipfftComplex REF(b)){
	hipfftComplex result = { a - b.x, -b.y };
	return result;
}

//multiply complex numbers
HOSTDEVICE hipfftComplex operator*(const hipfftComplex REF(a),
                                  const hipfftComplex REF(b)){
	hipfftComplex result = { a.x * b.x - a.y * b.y,
	                        a.y * b.x + a.x * b.y };
	return result;
}

//multiply complex with scalar
HOSTDEVICE hipfftComplex operator*(const hipfftComplex REF(a),
                                  const float REF(b)){
	hipfftComplex result = { a.x * b, a.y * b };
	return result;
}

//multiply scalar with complex
HOSTDEVICE hipfftComplex operator*(const float REF(a),
                                  const hipfftComplex REF(b)){
	hipfftComplex result = { a * b.x, a * b.y };
	return result;
}

//divide complex numbers
HOSTDEVICE hipfftComplex operator/(const hipfftComplex REF(a),
                                  const hipfftComplex REF(b)){
	float tmp = ( b.x * b.x + b.y * b.y );
	hipfftComplex result = { (a.x * b.x + a.y * b.y ) / tmp,
	                        (a.y * b.x - a.x * b.y ) / tmp };
	return result;
}

//divide complex by scalar
HOSTDEVICE hipfftComplex operator/(const hipfftComplex REF(a),
                                  const float REF(b)){
	hipfftComplex result = { a.x / b, a.y / b };
	return result;
}

//divide scalar by complex
HOSTDEVICE hipfftComplex operator/(const float REF(a),
                                  const hipfftComplex REF(b)){
	float tmp = ( b.x * b.x + b.y * b.y );
	hipfftComplex result = { ( a * b.x ) / tmp, ( -a * b.y ) / tmp };
	return result;
}

//complex conjugate
HOSTDEVICE hipfftComplex operator~(const hipfftComplex REF(a)){
	hipfftComplex result = { a.x, -a.y };
	return result;
}

HOSTDEVICE hipfftComplex exp(const hipfftComplex REF(a)){
	//hipfftComplex i = {0, 1};
	float eax = (float) exp(a.x);
	hipfftComplex result = {eax*(float)cos(a.y), eax*(float)sin(a.y)};
	return result;
}
